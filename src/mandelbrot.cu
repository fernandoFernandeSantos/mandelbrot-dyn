#include "hip/hip_runtime.h"
/** @file histo-global.cu histogram with global memory atomics */

#include <png.h>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#include "complex.h"

/** data size */
#define H (16 * 1024)
#define W (16 * 1024)
#define MAX_DWELL 512
#define BS 256

#define CUT_DWELL (MAX_DWELL / 4)
#define IMAGE_PATH "./mandelbrot.png"

/** CUDA check macro */
#define cucheck(call) \
	{\
	hipError_t res = (call);\
	if(res != hipSuccess) {\
	const char* err_str = hipGetErrorString(res);\
	fprintf(stderr, "%s (%d): %s in %s", __FILE__, __LINE__, err_str, #call);	\
	exit(-1);\
	}\
	}

/** time spent in device */
double gpu_time = 0;

/** a useful function to compute the number of threads */
int divup(int x, int y) {
	return x / y + (x % y ? 1 : 0);
}

/** gets the color, given the dwell */
void dwell_color(int *r, int *g, int *b, int dwell);

/** save the dwell into a PNG file 
 @remarks: code to save PNG file taken from here
 (error handling is removed):
 http://www.labbookpages.co.uk/software/imgProc/libPNG.html
 */
void save_image(const char *filename, int *dwells, int w, int h) {
	png_bytep row;

	FILE *fp = fopen(filename, "wb");
	png_structp png_ptr = png_create_write_struct(PNG_LIBPNG_VER_STRING, 0, 0,
			0);
	png_infop info_ptr = png_create_info_struct(png_ptr);
	// exception handling
	setjmp(png_jmpbuf(png_ptr));
	png_init_io(png_ptr, fp);
	// write header (8 bit colour depth)
	png_set_IHDR(png_ptr, info_ptr, w, h, 8, PNG_COLOR_TYPE_RGB,
			PNG_INTERLACE_NONE, PNG_COMPRESSION_TYPE_BASE,
			PNG_FILTER_TYPE_BASE);
	// set title
	png_text title_text;
	title_text.compression = PNG_TEXT_COMPRESSION_NONE;
	title_text.key = const_cast<char*>("Title");
	title_text.text = const_cast<char*>("Mandelbrot set, per-pixel");
	png_set_text(png_ptr, info_ptr, &title_text, 1);
	png_write_info(png_ptr, info_ptr);

	// write image data
	row = (png_bytep) malloc(3 * w * sizeof(png_byte));
	for (int y = 0; y < h; y++) {
		for (int x = 0; x < w; x++) {
			int r, g, b;
			dwell_color(&r, &g, &b, dwells[y * w + x]);
			row[3 * x + 0] = (png_byte) r;
			row[3 * x + 1] = (png_byte) g;
			row[3 * x + 2] = (png_byte) b;
		}
		png_write_row(png_ptr, row);
	}
	png_write_end(png_ptr, NULL);

	fclose(fp);
	png_free_data(png_ptr, info_ptr, PNG_FREE_ALL, -1);
	png_destroy_write_struct(&png_ptr, (png_infopp) NULL);
	free(row);
}  // save_image

/** computes the dwell for a single pixel */
template<typename real_t>
__device__ int pixel_dwell(int w, int h, complex<real_t> cmin,
		complex<real_t> cmax, int x, int y) {
	complex<real_t> dc = cmax - cmin;
	real_t fx = (real_t) x / w;
	real_t fy = (real_t) y / h;
	complex<real_t> c = cmin + complex<real_t>(fx * dc.re, fy * dc.im);
	int dwell = 0;
	complex<real_t> z = c;
	while (dwell < MAX_DWELL && z.abs2() < 2 * 2) {
		z = z * z + c;
		dwell++;
	}
	return dwell;
}  // pixel_dwell

/** computes the dwells for Mandelbrot image 
 @param dwells the output array
 @param w the width of the output image
 @param h the height of the output image
 @param cmin the complex value associated with the left-bottom corner of the
 image
 @param cmax the complex value associated with the right-top corner of the
 image
 */
template<typename real_t>
__global__ void mandelbrot_k(int *dwells, int w, int h, complex<real_t> cmin,
		complex<real_t> cmax) {
	// complex value to start iteration (c)
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int dwell = pixel_dwell(w, h, cmin, cmax, x, y);
	dwells[y * w + x] = dwell;
}  // mandelbrot_k

/** gets the color, given the dwell (on host) */

void dwell_color(int *r, int *g, int *b, int dwell) {
	// black for the Mandelbrot set
	if (dwell >= MAX_DWELL) {
		*r = *g = *b = 0;
	} else {
		// cut at zero
		if (dwell < 0)
			dwell = 0;
		if (dwell <= CUT_DWELL) {
			// from black to blue the first half
			*r = *g = 0;
			*b = 128 + dwell * 127 / (CUT_DWELL);
		} else {
			// from blue to white for the second half
			*b = 255;
			*r = *g = (dwell - CUT_DWELL) * 255 / (MAX_DWELL - CUT_DWELL);
		}
	}
}  // dwell_color

int main(int argc, char **argv) {
	// allocate memory
	int w = W, h = H;
	size_t dwell_sz = w * h * sizeof(int);
	int *h_dwells, *d_dwells;
	cucheck(hipMalloc((void** )&d_dwells, dwell_sz));
	h_dwells = (int*) malloc(dwell_sz);

	// compute the dwells, copy them back
	double t1 = omp_get_wtime();
	dim3 bs(64, 4), grid(divup(w, bs.x), divup(h, bs.y));
	mandelbrot_k<<<grid, bs>>>(d_dwells, w, h, complex<double>(-1.5, -1),
			complex<double>(0.5, 1));
	cucheck(hipDeviceSynchronize());
	double t2 = omp_get_wtime();
	cucheck(hipMemcpy(h_dwells, d_dwells, dwell_sz, hipMemcpyDeviceToHost));
	gpu_time = t2 - t1;

	// save the image to PNG 
	save_image(IMAGE_PATH, h_dwells, w, h);

	// print performance
	printf("Mandelbrot set computed in %.3lf s, at %.3lf Mpix/s\n", gpu_time,
			h * w * 1e-6 / gpu_time);

	// free data
	hipFree(d_dwells);
	free(h_dwells);
	return 0;
}  // main
